#include "hip/hip_runtime.h"
//extern "C" {
#include <cudax.h>
//}
//#include <common.h>

#define MAX_CELL_PER_THREAD 2

/*
__device__ uint8_t countAliveCells(uint8_t *matrix, uint32_t x0, uint32_t x1, uint32_t x2, uint32_t y0, uint32_t y1, uint32_t y2) {
    return matrix[x0 + y0] + matrix[x1 + y0] + matrix[x2 + y0] + matrix[x0 + y1]
         + matrix[x2 + y1] + matrix[x0 + y2] + matrix[x1 + y2] + matrix[x2 + y2];
}*/

GenStateGpu_p create_gen_gpu(uint32_t rows, uint32_t cols){
    uint32_t size = rows * cols * sizeof(uint8_t);
    uint8_t *matrix = NULL; 
    hipError_t err = hipMalloc(&matrix, size);
    if (err != hipSuccess){
        fprintf(stderr, "Failed to allocate d_Input (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    GenStateGpu_p gen = (GenStateGpu_p) malloc(sizeof(GenStateGpu));
    gen->matrix = matrix;
    gen->rows = rows;
    gen->cols = cols;
    return gen;
}

void gen_h2d(GenState_p gen_host, GenStateGpu_p gen_device){
    if (gen_host->rows == gen_device->rows && gen_host->cols == gen_device->cols){
        hipError_t err = hipMemcpy(gen_device->matrix, gen_host->matrix, sizeof(uint8_t)*gen_host->rows * gen_host->cols, hipMemcpyHostToDevice);//dest, sorg, size, kind
        
        if (err != hipSuccess){
        fprintf(stderr, "Failed to copy GenState from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
        }
    }
}

void gen_d2h(GenStateGpu_p gen_device, GenState_p gen_host){
    if (gen_host->rows == gen_device->rows && gen_host->cols == gen_device->cols){
        hipError_t err = hipMemcpy(gen_host->matrix, gen_device->matrix, sizeof(uint8_t)*gen_host->rows * gen_host->cols, hipMemcpyDeviceToHost);//dest, sorg, size, kind
        
        if (err != hipSuccess){
            fprintf(stderr, "Failed to copy GenState device to host (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }
    }
}

void free_gen_gpu(GenStateGpu_p gen){
    hipFree(gen->matrix);
    free(gen);
}

__device__ void gpu_swap(void **a, void **b) {
    void *temp = *a;
    *a = *b;
    *b = temp;
}

/**
 * 1) Kernel device routine
 */
__global__ void kernel_compute_gen_singleblock_1(uint8_t *matrix_src, uint8_t *matrix_dst,  uint32_t dim_m1, uint32_t cols, uint32_t iterations) {
    //each thread loads one element from global to shared mem
    uint32_t cell = threadIdx.x;
    
    uint32_t x1 = cell     & cols-1; //% cols;
    uint32_t x0 = (x1 - 1) & cols-1; //% cols;
    uint32_t x2 = (x1 + 1) & cols-1; //% cols;

    uint32_t y1 = cell - x1;
    uint32_t y0 = (y1 - cols) & dim_m1; //% dim;
    uint32_t y2 = (y1 + cols) & dim_m1; //% dim;

    for (uint32_t iter = 0; iter < iterations; iter++){
        uint8_t aliveCells = matrix_src[x0 + y0] + matrix_src[x1 + y0] + matrix_src[x2 + y0] + matrix_src[x0 + y1] +
                                    matrix_src[x2 + y1] + matrix_src[x0 + y2] + matrix_src[x1 + y2] + matrix_src[x2 + y2];
        matrix_dst[y1 + x1] = (aliveCells == 3 || (aliveCells == 2 && matrix_src[y1 + x1])) ? 1 : 0; 
        gpu_swap((void**)&matrix_dst,(void**)&matrix_src);
        __syncthreads();
    } 
}

/**
 * 2) Kernel device routine
 */
 __global__ void kernel_compute_gen_singleblock(uint8_t *matrix_src, uint8_t *matrix_dst,  uint32_t dim_m1, uint32_t cols, uint32_t iterations, uint32_t cellPerThreads) {
    //each thread loads one element from global to shared mem
    uint32_t start = threadIdx.x * cellPerThreads; //punto di partenza di ogni thread, una cella ogni 32
    uint32_t end =  cellPerThreads + start;

    for (uint32_t iter = 0; iter < iterations; iter++){
        for (uint32_t cell = start; cell < end; cell++){
            uint32_t x1 = cell     & cols-1; //% cols;
            uint32_t x0 = (x1 - 1) & cols-1; //% cols;
            uint32_t x2 = (x1 + 1) & cols-1; //% cols;

            uint32_t y1 = cell - x1;
            uint32_t y0 = (y1 - cols) & dim_m1; //% dim;
            uint32_t y2 = (y1 + cols) & dim_m1; //% dim;

            uint8_t aliveCells = matrix_src[x0 + y0] + matrix_src[x1 + y0] + matrix_src[x2 + y0] + matrix_src[x0 + y1] +
                                 matrix_src[x2 + y1] + matrix_src[x0 + y2] + matrix_src[x1 + y2] + matrix_src[x2 + y2];
        
            matrix_dst[y1 + x1] = (aliveCells == 3 || (aliveCells == 2 && matrix_src[y1 + x1])) ? 1 : 0; 
        }

        gpu_swap((void**)&matrix_dst,(void**)&matrix_src);
        __syncthreads();
    } 
}

/**
 * 3) Kernel device routine
 */
 __global__ void kernel_compute_gen_multiblocks(uint8_t *matrix_src, uint8_t *matrix_dst,  uint32_t dim_m1, uint32_t cols, uint32_t cellPerThreads) {
    //each thread loads one element from global to shared mem
    //uint32_t dim = rows * cols;    
    //if(threadIdx.x == 0)
    //  printf("Il mio start vale %d mentre il mio blockid vale %d\n", start, blockIdx.x);
    
    //punto di partenza di ogni thread, una cella ogni 32
    uint32_t start = (blockIdx.x*blockDim.x + threadIdx.x) * cellPerThreads;
    uint32_t end =  cellPerThreads + start;

    for (uint32_t cell = start; cell < end; cell++){
        uint32_t x1 = cell     & cols-1; //% cols;
        uint32_t x0 = (x1 - 1) & cols-1; //% cols;
        uint32_t x2 = (x1 + 1) & cols-1; //% cols;

        uint32_t y1 = cell - x1;
        uint32_t y0 = (y1 - cols) & dim_m1; //% dim;
        uint32_t y2 = (y1 + cols) & dim_m1; //% dim;

        uint8_t aliveCells = matrix_src[x0 + y0] + matrix_src[x1 + y0] + matrix_src[x2 + y0] + matrix_src[x0 + y1] +
                                matrix_src[x2 + y1] + matrix_src[x0 + y2] + matrix_src[x1 + y2] + matrix_src[x2 + y2];
    
        matrix_dst[y1 + x1] = (aliveCells == 3 || (aliveCells == 2 && matrix_src[y1 + x1])) ? 1 : 0; 
    }     
}

uint32_t getDeviceInfo(){
    int deviceCount = 0;
    hipSetDevice(deviceCount);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, deviceCount);
    return deviceProp.maxThreadsPerBlock;
}

void compute_generation_on_gpu(GenStateGpu_p s1, GenStateGpu_p s2, uint32_t iterations){
    // Load gpu info for optimal load balancing   
    uint32_t dim_world = s1->rows * s1->cols;
    uint32_t threadsPerBlock = getDeviceInfo();
    
    if (dim_world <= threadsPerBlock){
        //printf("1 if");
        kernel_compute_gen_singleblock_1<<<1, dim_world>>>(s1->matrix, s2->matrix, s1->rows, s1->cols, iterations);//num_block, dim_block,  
    } else {
        
        uint32_t cellPerThreads = dim_world / threadsPerBlock; 
        if (cellPerThreads <= MAX_CELL_PER_THREAD){ 
        
            //kernel con un unico blocco con la barrier interna
            //printf("2 if");
            kernel_compute_gen_singleblock<<<1, threadsPerBlock>>>(s1->matrix, s2->matrix, s1->rows, s1->cols, iterations, cellPerThreads);//num_block, dim_block,          
        } else {//se sono più di 32 celle per thread si spalma il lavoro su più blocchi
            //printf("3 if");
            uint32_t totalBlocks = cellPerThreads / MAX_CELL_PER_THREAD;//numero di blocchi sarà sempre potenza di 2 positiva 
            for (uint32_t iter = 0; iter< iterations; iter++){
        
                kernel_compute_gen_multiblocks<<<totalBlocks, threadsPerBlock>>>(s1->matrix, s2->matrix, dim_world-1, s1->cols, MAX_CELL_PER_THREAD);//num_block, dim_block,          
                swap((void **) &s1, (void **) &s2);
        
            }
        }
    }
}

/*
BACKUP void compute_generation_on_gpu(GenStateGpu_p s1, GenStateGpu_p s2, uint32_t iterations){
    // Load gpu info for optimal load balancing   
    uint32_t dim_world = s1->rows * s1->cols;
    uint32_t threadsPerBlock = getDeviceInfo();
    
    if (dim_world <= threadsPerBlock){
        kernel_compute_gen<<<1, dim_world>>>(s1->matrix, s2->matrix, s1->rows, s1->cols, iterations);//num_block, dim_block,  
    } else {
        uint32_t totalThreads = dim_world / 32; 
        if ( totalThreads <= threadsPerBlock){
            //kernel con un unico blocco con la barrier interna
            kernel_compute_gen_32<<<1, totalThreads>>>(s1->matrix, s2->matrix, s1->rows, s1->cols, iterations);//num_block, dim_block,          
        }/* else {
            uint32_t totalBlocks = totalThreads / threadsPerBlock; 
            for (uint32_t i = 0; i< iterations; i++)
            //swap((void *) &s1, (void *) &s2);
        }
    }
}*/

 
void compute_cpu_generations_on_gpu(GenState_p s1, GenState_p s2, uint32_t iterations){
    GenStateGpu_p gen_device_1 = create_gen_gpu(s1->rows, s1->cols);
    GenStateGpu_p gen_device_2 = create_gen_gpu(s1->rows, s1->cols);

    gen_h2d(s1, gen_device_1);
    compute_generation_on_gpu(gen_device_1, gen_device_2, iterations);
    gen_d2h(gen_device_2, s2);
    
    // FINE KERNEL
    free_gen_gpu(gen_device_1);
    free_gen_gpu(gen_device_2);
}




/*
void game_Gpu(uint32_t rows, uint32_t cols, uint32_t iterations){
    if (isPow2(rows) && isPow2(rows)){
        GenState_p gen_host = create_gen(rows, cols);
        random_gen(gen_host);
        GenStateGpu_p gen_device_1 = create_gen_gpu(rows, cols);
        GenStateGpu_p gen_device_2 = create_gen_gpu(rows, cols);
        gen_h2d(gen_host, gen_device_1);
        compute_generation_on_gpu(gen_device_1, gen_device_2, iterations);
        
        // FINE KERNEL
        free_gen(gen_host);
        free_gen_gpu(gen_device_1);
        free_gen_gpu(gen_device_2);
    } else {
        puts("Rows or Cols are not a power of 2!");
    }
}*/

/*
 * Host main routine
 */
 /*
int main(void){
    //game_Gpu(32,32, 1000);
    printf("eseguito cuda");
}*/
